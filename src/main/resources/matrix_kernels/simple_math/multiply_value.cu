
#include <hip/hip_runtime.h>
extern "C"
__global__ void multiply_value(int n, double *a, double b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; //output x
    if (i < n)
    {
    	c[i] = a[i] * b;
    }
}